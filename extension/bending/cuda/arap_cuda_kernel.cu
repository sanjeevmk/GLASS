#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../eigen/Eigen/Core"
#include "../eigen/Eigen/SVD"
#include "../eigen/Eigen/Eigen"
#include "svd3_cuda.h"
using namespace Eigen;

__global__
void ArapRotationKernel(
    const int b, const int n,
    const float* xyz,
    const float* xyzP,
    int* neighborList,
    int* numNeighbors,
    int* accnumNeighbors,
    const float* weightMatrix,
    const int blocklength,
    float* rotations,
    float* updated_rotations)
{
	for(int bind=blockIdx.x;bind<b;bind+=gridDim.x){
        for(int indi=threadIdx.x+blockIdx.y*blockDim.x;indi<n;indi+=blockDim.x*gridDim.y){
        //for(int indi=blockDim.x*blockIdx.x+threadIdx.x;indi<n;indi+=blockDim.x*gridDim.x){
            int gradIndex = bind*n*3 + indi*3;
            int index = bind*n*3 + indi*3;

            int neighIndex = bind*n + indi;
            int nNeighbors = numNeighbors[neighIndex];

            int accneighborsIndex = bind*n + indi;
            int numAccNeighbors = accnumNeighbors[accneighborsIndex];

            int neighListIndex = bind*blocklength + numAccNeighbors;
            int weightMatrixIndex = bind*blocklength + numAccNeighbors;

            float x1 = xyz[index] ; float y1 = xyz[index+1] ; float z1 = xyz[index+2] ;
            float x1P,y1P,z1P;
            x1P = xyzP[index] ; y1P = xyzP[index+1] ; z1P = xyzP[index+2] ;
            MatrixXf Pi(3,nNeighbors);
            MatrixXf PiP(nNeighbors,3);
            MatrixXf wij = MatrixXf::Zero(nNeighbors,nNeighbors);
            MatrixXf Ri(3,3);
            MatrixXf Rj(3,3);

            MatrixXf neighborRotTerms = MatrixXf::Zero(3,3);
            for(int nIndex=0;nIndex<nNeighbors;nIndex++){
                int neighborPointIndex = neighborList[neighListIndex+nIndex];
                int index2 = bind*n*3 + neighborPointIndex*3;
                float x2 = xyz[index2] ; float y2 = xyz[index2+1] ; float z2 = xyz[index2+2] ; 
                float x2P,y2P,z2P;
                x2P = xyzP[index2] ; y2P = xyzP[index2+1] ; z2P = xyzP[index2+2] ; 
                Pi(0,nIndex) = x1-x2 ; Pi(1,nIndex) = y1-y2 ; Pi(2,nIndex) = z1-z2 ;
                wij(nIndex,nIndex) =  weightMatrix[weightMatrixIndex+nIndex];
                PiP(nIndex,0) = x1P - x2P ; PiP(nIndex,1) = y1P - y2P ; PiP(nIndex,2) = z1P - z2P ;

                int neighRotIndex = bind*n*9 + neighborPointIndex*9;
                Rj(0,0) = rotations[neighRotIndex] ;  Rj(0,1) = rotations[neighRotIndex+1] ; Rj(0,2) = rotations[neighRotIndex+2] ;
                Rj(1,0) = rotations[neighRotIndex+3] ; Rj(1,1) = rotations[neighRotIndex+4] ; Rj(1,2) = rotations[neighRotIndex+5] ;
                Rj(2,0) = rotations[neighRotIndex+6] ; Rj(2,1) = rotations[neighRotIndex+7] ; Rj(2,2) = rotations[neighRotIndex+8] ;

                MatrixXf _neighRotTerm = 2.0 * wij(nIndex,nIndex) * Rj.transpose();
                neighborRotTerms += _neighRotTerm;
            }

            MatrixXf Si = ((Pi*wij)*PiP) + neighborRotTerms;
            float u00,u01,u02,u10,u11,u12,u20,u21,u22;
            float v00,v01,v02,v10,v11,v12,v20,v21,v22;
            float s0,s1,s2;
            svd(Si(0,0),Si(0,1),Si(0,2),Si(1,0),Si(1,1),Si(1,2),Si(2,0),Si(2,1),Si(2,2),u00,u01,u02,u10,u11,u12,u20,u21,u22,s0,s1,s2,v00,v01,v02,v10,v11,v12,v20,v21,v22);
			MatrixXf U(3,3);
            MatrixXf V(3,3);
            U(0,0) = u00; U(0,1) = u01 ; U(0,2) = u02;
            U(1,0) = u10; U(1,1) = u11 ; U(1,2) = u12;
            U(2,0) = u20; U(2,1) = u21 ; U(2,2) = u22;
            V(0,0) = v00; V(0,1) = v01 ; V(0,2) = v02;
            V(1,0) = v10; V(1,1) = v11 ; V(1,2) = v12;
            V(2,0) = v20; V(2,1) = v21 ; V(2,2) = v22;
            Ri = V*U.transpose();
            float determinant = Ri(0,0)*(Ri(1,1)*Ri(2,2) - Ri(1,2)*Ri(2,1)) - Ri(0,1)*(Ri(1,0)*Ri(2,2) - Ri(1,2)*Ri(2,0)) + Ri(0,2)*(Ri(1,0)*Ri(2,1) - Ri(1,1)*Ri(2,0)) ;
            if(determinant<0){
                U.col(2)*=-1;
                Ri = V*U.transpose();
            }
            int rotIndex = bind*n*9 + indi*9;
            updated_rotations[rotIndex] = Ri(0,0) ; updated_rotations[rotIndex+1] = Ri(0,1) ; updated_rotations[rotIndex+2] = Ri(0,2) ;
            updated_rotations[rotIndex+3] = Ri(1,0) ; updated_rotations[rotIndex+4] = Ri(1,1) ; updated_rotations[rotIndex+5] = Ri(1,2) ;
            updated_rotations[rotIndex+6] = Ri(2,0) ; updated_rotations[rotIndex+7] = Ri(2,1) ; updated_rotations[rotIndex+8] = Ri(2,2) ;
        }
    }
}

void ArapRotationKernelLauncher(
    const int b, const int n,
    const float* xyz,
    const float* xyzP,
    int* neighborList,
    int* numNeighbors,
    int* accnumNeighbors,
    const float* weightMatrix,
    const int blocklength,
    float* rotations,
	float* updated_rotations)
{
	ArapRotationKernel<<<dim3(b,(n+7)/8),8>>>(b, n, xyz, xyzP,neighborList,numNeighbors,accnumNeighbors,weightMatrix,blocklength,rotations,updated_rotations);
	//ArapGradKernel2D<<<dim3(32,16,1),512>>>(b, n, xyz, xyzP,neighborList,numNeighbors,weightMatrix,rotations,gradxyz);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	    printf("error in arap grad computation: %s\n", hipGetErrorString(err));
}
